#include "hip/hip_runtime.h"
#include <cv_bridge/cv_bridge.h>
#include <gum_perception/sam_publisher.h>

#include <gum/perception/feature/outlier_rejection.h>
#include <gum/perception/utils/utils.cuh>
#include <gum/utils/cuda_utils.cuh>

namespace gum {
namespace perception {
SAMPublisher::SAMPublisher(const std::string &node_name,
                           const std::string &color_topic,
                           const std::string &depth_topic,
                           const std::string &sam_topic,
                           const std::string &sam_encoder_checkpoint,
                           const std::string &sam_decoder_checkpoint,
                           const std::string &superpoint_checkpoint,
                           const std::string &lightglue_checkpoint,
                           const std::string &ostrack_checkpoint,
                           const std::string &trt_engine_cache_path)
    : rclcpp::Node(node_name) {
  this->declare_parameter("device", rclcpp::PARAMETER_INTEGER);
  this->declare_parameter("height", rclcpp::PARAMETER_INTEGER);
  this->declare_parameter("width", rclcpp::PARAMETER_INTEGER);
  this->declare_parameter("intrinsics", rclcpp::PARAMETER_DOUBLE_ARRAY);
  this->declare_parameter("depth_scale", rclcpp::PARAMETER_DOUBLE);

  m_color_subscriber =
      std::make_shared<message_filters::Subscriber<sensor_msgs::msg::Image>>(
          this, color_topic);
  m_depth_subscriber =
      std::make_shared<message_filters::Subscriber<sensor_msgs::msg::Image>>(
          this, depth_topic);
  m_synchronizer =
      std::make_shared<message_filters::Synchronizer<ApproximatePolicy>>(
          ApproximatePolicy(10), *m_color_subscriber, *m_depth_subscriber);
  m_synchronizer->registerCallback(&SAMPublisher::CallBack, this);
  m_sam = std::make_shared<gum::perception::segmentation::SAM>(
      sam_encoder_checkpoint, sam_decoder_checkpoint);
  m_superpoint = std::make_shared<gum::perception::feature::SuperPoint>(
      superpoint_checkpoint, trt_engine_cache_path);
  m_lightglue = std::make_shared<gum::perception::feature::LightGlue>(
      lightglue_checkpoint, trt_engine_cache_path);

  m_device = this->get_parameter("device").as_int();
  m_height = this->get_parameter("height").as_int();
  m_width = this->get_parameter("width").as_int();
  m_depth_scale = this->get_parameter("depth_scale").as_double();
  auto intrinsics = this->get_parameter("intrinsics").as_double_array();

  m_dataset = std::make_shared<gum::perception::dataset::RealSenseDataset<
      gum::perception::dataset::Device::GPU>>(
      m_device, m_height, m_width, intrinsics[0], intrinsics[1], intrinsics[2],
      intrinsics[3], intrinsics[4], intrinsics[5], intrinsics[6], intrinsics[7],
      m_depth_scale);
}

void SAMPublisher::Process(const Frame &prev_frame, Frame &curr_frame) {
  std::vector<Eigen::Vector2f> initial_keypoints_v;
  std::vector<float> initial_keypoint_scores_v;
  std::vector<Eigen::Vector<float, 256>> initial_descriptors_v;
  std::vector<Eigen::Vector2f> initial_normalized_keypoints_v;

  const Eigen::Vector4f init_bbox;

  torch::Tensor extended_mask_cpu = torch::empty(
      prev_frame.mask_cpu.sizes(),
      torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCPU));
  int extended_radius = std::round(
      std::max(1.f, 0.080f * std::sqrt((init_bbox[3] - init_bbox[1]) *
                                       (init_bbox[2] - init_bbox[0]))));
  int shrinked_radius = 1;
  gum::perception::utils::ExtendMasks(m_height, m_width, curr_frame.bbox,
                                      prev_frame.mask_cpu.data_ptr<uint8_t>(),
                                      extended_mask_cpu.data_ptr<uint8_t>(),
                                      extended_radius);
  torch::Tensor shrinked_mask_cpu = torch::empty(
      prev_frame.mask_cpu.sizes(),
      torch::TensorOptions().dtype(torch::kUInt8).device(torch::kCPU));
  gum::perception::utils::ShrinkMasks(m_height, m_width, prev_frame.bbox,
                                      prev_frame.mask_cpu.data_ptr<uint8_t>(),
                                      shrinked_mask_cpu.data_ptr<uint8_t>(),
                                      shrinked_radius);

  cv::Mat initial_cropped_image;
  curr_frame
      .image(cv::Range(curr_frame.bbox[1], curr_frame.bbox[3]),
             cv::Range(curr_frame.bbox[0], curr_frame.bbox[2]))
      .copyTo(initial_cropped_image,
              cv::Mat(curr_frame.image.size(), CV_8U,
                      extended_mask_cpu.data_ptr<uint8_t>())(
                  cv::Range(curr_frame.bbox[1], curr_frame.bbox[3]),
                  cv::Range(curr_frame.bbox[0], curr_frame.bbox[2])));

  // Initial Mask
  cv::cvtColor(initial_cropped_image, initial_cropped_image,
               cv::COLOR_RGB2GRAY);

  // Extract Keypoints
  m_superpoint->Extract(initial_cropped_image, initial_keypoints_v,
                        initial_normalized_keypoints_v,
                        initial_keypoint_scores_v, initial_descriptors_v);
  for (auto &initial_keypoint : initial_keypoints_v) {
    initial_keypoint += curr_frame.offset;
  }

  int num_initial_keypoints = initial_keypoints_v.size();
  gum::perception::utils::SelectKeyPointsByDepth(
      num_initial_keypoints, m_min_depth, m_max_depth, m_depth_scale,
      curr_frame.depth, initial_keypoints_v, initial_descriptors_v,
      initial_normalized_keypoints_v, curr_frame.keypoints_v,
      curr_frame.descriptors_v, curr_frame.normalized_keypoints_v);

  // Point Clouds
  int num_keypoints = curr_frame.keypoints_v.size();
  gum::perception::utils::GetPointClouds(
      num_keypoints, m_intrinsics[0], m_intrinsics[1], m_intrinsics[2],
      m_intrinsics[3], m_depth_scale, curr_frame.depth, curr_frame.keypoints_v,
      curr_frame.point_clouds_v);

  // Feature Matching
  thrust::device_vector<int> d_initial_matches_v;
  std::vector<float> match_scores_v;
  std::vector<Eigen::Vector2i> initial_matches_v;
  m_lightglue->Match(prev_frame.normalized_keypoints_v,
                     curr_frame.normalized_keypoints_v,
                     prev_frame.descriptors_v, curr_frame.descriptors_v,
                     initial_matches_v, match_scores_v);
  int num_initial_matches = initial_matches_v.size();
  d_initial_matches_v.resize(2 * num_initial_matches);
  gum::utils::HostArrayOfMatrixToDeviceMatrixOfArray(initial_matches_v,
                                                     d_initial_matches_v);
  thrust::device_vector<int> d_matches_v;
  gum::perception::feature::RejectOutliers(
      *m_handle, m_graph_params, m_leiden_params, m_outlier_tolerance,
      prev_frame.point_clouds_v.size() / 3,
      curr_frame.point_clouds_v.size() / 3, num_initial_matches,
      prev_frame.point_clouds_v, curr_frame.point_clouds_v, d_initial_matches_v,
      d_matches_v);
  int num_matches = d_matches_v.size() / 2;
  Eigen::Matrix<float, 3, 4> relative_pose;
  gum::perception::feature::EstimateRelativePose(
      *m_handle, prev_frame.point_clouds_v.size() / 3,
      curr_frame.point_clouds_v.size() / 3, num_matches,
      prev_frame.point_clouds_v, curr_frame.point_clouds_v, d_matches_v,
      relative_pose);

  // Segmentation
  std::vector<Eigen::Vector2i> matches_v(num_matches);
  gum::utils::DeviceMatrixOfArrayToHostArrayOfMatrix(d_matches_v, matches_v);
  std::vector<int> selected_match_indices_v;
  std::vector<Eigen::Vector2f> point_coords_v;
  gum::perception::utils::SelectMatchesForSAM(
      m_height, m_width, prev_frame.keypoints_v.size(),
      curr_frame.keypoints_v.size(), num_matches, prev_frame.bbox.cast<float>(),
      curr_frame.bbox.cast<float>(), prev_frame.mask_cpu.data_ptr<uint8_t>(),
      prev_frame.mask_cpu.data_ptr<uint8_t>(), prev_frame.keypoints_v,
      curr_frame.keypoints_v, matches_v, selected_match_indices_v,
      point_coords_v);
  std::vector<float> point_labels_v(point_coords_v.size(), 1.0f);

  torch::Tensor masks, scores, logits;
  m_sam->SetImage(curr_frame.image);
  m_sam->Query(point_coords_v, point_labels_v, init_bbox, masks, scores,
               logits);
  curr_frame.mask_gpu = masks[0][1].to(torch::kUInt8);
  curr_frame.mask_cpu = curr_frame.mask_gpu.to(torch::kCPU);
  gum::perception::utils::FilterMaskByDepth(
      m_height, m_width, curr_frame.bbox, m_min_depth, m_max_depth,
      m_depth_scale, curr_frame.depth, curr_frame.mask_cpu.data_ptr<uint8_t>());
  gum::perception::utils::RefineMask(m_height, m_width, curr_frame.bbox,
                                     curr_frame.mask_cpu.data_ptr<uint8_t>());
  curr_frame.mask_gpu = curr_frame.mask_cpu.to(curr_frame.mask_gpu.device());
}

void SAMPublisher::CallBack(
    const sensor_msgs::msg::Image::ConstSharedPtr &color_msg,
    const sensor_msgs::msg::Image::ConstSharedPtr &depth_msg) {
  cv_bridge::CvImagePtr color_ptr, depth_ptr;
  color_ptr =
      cv_bridge::toCvCopy(color_msg, sensor_msgs::image_encodings::BGR8);
  depth_ptr =
      cv_bridge::toCvCopy(depth_msg, sensor_msgs::image_encodings::MONO16);
  double timestamp = double(color_msg->header.stamp.sec) +
                     1e-9 * double(color_msg->header.stamp.nanosec);
  m_dataset->AddFrame(
      {timestamp, std::move(color_ptr->image), std::move(depth_ptr->image)});

  if (m_dataset->GetNumFrames() >= 1000) {
    m_dataset->Clear();
  }
}
} // namespace perception
} // namespace gum